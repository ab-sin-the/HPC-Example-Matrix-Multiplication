// GPU with parallelization version
// Parallelization is implemented with CUDA

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <hip/hip_runtime.h>
using namespace std;

// __global__ means the function runs on GPU, and called from CPU (in this case the function is called by main(), which runs on CPU)
__global__
void multiply_matrix(int n, int* matrixA, int* matrixB, int* matrix_res) {
    int line_num = blockIdx.x * blockDim.x + threadIdx.x;
    if (line_num < n) {
        for (int i = 0; i < n; i ++) {
            for (int j = 0; j < n; j ++) {
                //
                // matrix_res[line_num][i] = sum(j) matrixA[line_num][j] * matrixB[j][i]
                //
                matrix_res[line_num * n + i] += matrixA[line_num * n + j] * matrixB[j * n + i];
            }
        }
    }
}

int main(int argc, char const *argv[])
{   
    if (argc < 2) {
        cout << "Wrong number of arguments!!" << endl;
        return -1;
    }
    int n = atoi(argv[1]);
    //
    // We will next generate two matrices both of size n*n with random int number with range [1, 512]
    // We will also generate a zero matrix to store the result of multiplication
    // For now, you do not need to understand the "new" expression
    //

    int *matrixA = new int[n*n]; 
    int *matrixB = new int[n*n];
    int *matrix_res = new int[n*n];
    int *dMatrixA;
    int *dMatrixB;
    int *dMatrix_res;
    // Notice that we use different index method
    // We store all the data of the matrix in one row, so matrix[i * n + j] is previous matrix[i][j]
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            matrixA[i * n + j] = (rand() % 512) + 1;
            matrixB[i * n + j] = (rand() % 512) + 1;
            matrix_res[i * n + j] = 0;
        }
    }

    //
    // We will next multiply these two matrices (matrixA * matrixB) and count the time
    // The result of multiplication will be stored in matrix_res
    //

    clock_t start = clock();

    //
    // Since CPU and GPU use different memory, we need to allocate memory on GPU
    // We use id dMatrix to mean Matrix stored in device
    //

    hipMallocManaged(&dMatrixA, n * n * sizeof(int));
    hipMallocManaged(&dMatrixB, n * n * sizeof(int));
    hipMallocManaged(&dMatrix_res, n * n * sizeof(int));

    hipMemcpy(dMatrixA, matrixA, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dMatrixB, matrixB, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dMatrix_res, matrix_res, n * n * sizeof(int), hipMemcpyHostToDevice);

    //
    // The code below call the function run on GPU and wait it to finish
    // Each thread will calculate one line
    //

    multiply_matrix<<< ((n + 255) / 256) , 256>>> (n, dMatrixA, dMatrixB, dMatrix_res);
    hipDeviceSynchronize();

    
    hipMemcpy(matrix_res, dMatrix_res, n * n * sizeof(int), hipMemcpyDeviceToHost);
    cout << "It takes " << (clock() - start) / (double) CLOCKS_PER_SEC  << " seconds to multiply two matrices with size " << n << " * " << n << endl;
    //
    // Clean up the memory, you do not need to understand this part
    //
    
    hipFree(dMatrixA);
    hipFree(dMatrixB);
    hipFree(dMatrix_res);
    delete [] matrixA;
    delete [] matrixB;
    delete [] matrix_res;

    return 0;
}

